#include "hip/hip_runtime.h"
#include "common.h"
#include "mmio.h"
#include "read_mtx.h"
#include "tranpose.h"
#include "YYSpTRSV.h"



int main(int argc, char ** argv)
{

    // report precision of floating-point
    printf("---------------------------------------------------------------------------------------------\n");
    char  *precision;
    if (sizeof(VALUE_TYPE) == 4)
    {
        precision = (char *)"32-bit Single Precision";
    }
    else if (sizeof(VALUE_TYPE) == 8)
    {
        precision = (char *)"64-bit Double Precision";
    }
    else
    {
        printf("Wrong precision. Program exit!\n");
        return 0;
    }
    
    printf("PRECISION = %s\n", precision);
    printf("Benchmark REPEAT = %i\n", BENCH_REPEAT);
    printf("---------------------------------------------------------------------------------------------\n");
    
    int m, n, nnzA;
    int *csrRowPtrA;
    int *csrColIdxA;
    VALUE_TYPE *csrValA;
    
    //ex: ./YYSpTRSV webbase-1M.mtx
    int argi = 1;
    
    char  *filename;
    if(argc > argi)
    {
        filename = argv[argi];
        argi++;
    }
    printf("-------------- %s --------------\n", filename);
    
    read_mtx(filename, &m, &n, &nnzA, &csrRowPtrA, &csrColIdxA, &csrValA);

    //printf("read_mtx finish\n");
    
    /* extract Matrix L with the unit-lower triangular sparsity structure of input Matrix A */
    int nnzL = 0;
    int *csrRowPtrL_tmp ;
    int *csrColIdxL_tmp ;
    VALUE_TYPE *csrValL_tmp;
    if(m<=n)
        n=m;
    else
        m=n;
    if (m<1)
        return 0;

    change2tran(m, nnzA,csrRowPtrA, csrColIdxA, csrValA, &nnzL, &csrRowPtrL_tmp, &csrColIdxL_tmp, &csrValL_tmp);
    printf("A's unit-lower triangular L: ( %i, %i ) nnz = %i\n", m, n, nnzL);

    
    free(csrColIdxA);
    free(csrValA);
    free(csrRowPtrA);

    if(m==0 || nnzL==0)
        return -3;
    

    /* calculate the number of layer and parallelism of matrix L */
    int layer;
    double parallelism;

    matrix_layer(m,n,nnzL,csrRowPtrL_tmp,csrColIdxL_tmp,csrValL_tmp,&layer,&parallelism);

    /* get vector b and reference x for Lx=b */
    VALUE_TYPE *x_ref;
    VALUE_TYPE *b;
    get_x_b(m, n, csrRowPtrL_tmp, csrColIdxL_tmp, csrValL_tmp, &x_ref, &b);
    
    VALUE_TYPE *x = (VALUE_TYPE *)malloc(sizeof(VALUE_TYPE) * n);
    
    /* The border between thread-level and warp-level algorithms, according to the number of non-zero elements in each row of the matrix L*/
    int border = 10;
    
    
    /* !!!!!! start computing SpTRSV !!!!!!!! */
    double solve_time,gflops,bandwith,pre_time,warp_occupy,element_occupy;
    int success = YYSpTRSV_csr(m,n,nnzL,csrRowPtrL_tmp, csrColIdxL_tmp, csrValL_tmp, b, x, border, &solve_time, &gflops, &bandwith, &pre_time, &warp_occupy, &element_occupy);
    
    
    /* check solution x */
    int err_counter = 0;
    for (int i = 0; i < n; i++)
    {
        if (abs(x_ref[i] - x[i]) > 0.01 * abs(x_ref[i]))
        {
            err_counter++;
        }
    }
    
    if (!err_counter)
        printf("YYSpTRSV on L passed!\n");

    
    printf("The unit-lower triangular L (%s): ( %i, %i ) nnz = %i, layer = %d, parallelism = %4.2f\n", filename, m, n, nnzL, layer, parallelism);
    printf("The preprocessing time = %4.2f ms, solving time =  %4.2f ms, throught = %4.2f gflops, bandwidth = %4.2f GB/s.\n", pre_time, solve_time, gflops, bandwith);
    


    free(x);
    free(x_ref);
    free(b);
    free(csrColIdxL_tmp);
    free(csrValL_tmp);
    free(csrRowPtrL_tmp);
    
    return 0;
}
    

